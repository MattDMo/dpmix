
#include <hip/hip_runtime.h>

__global__ void
apply_rows_max_cm(float* X, /** matrix to apply ... column major **/
	               	      float* y, /** result vector  **/
			      int rows,
			      int cols
  ) {

  unsigned int thidx = threadIdx.x;
  unsigned int thidy = threadIdx.y;
  unsigned int bid = blockIdx.x;
  unsigned int bdx = blockDim.x; // assumed equal to blockDim.y .. 16 or 32 ..

  int currow = bdx*bid;

  // flexible block size 
  extern __shared__ float shared_data[];
  float *sh_max = shared_data + bdx*bdx;
  
  float cur_val;
  for(int chunk = 0; chunk < cols; chunk+=bdx){
  	  // get some values chunking accross rows ...
	  shared_data[thidy*bdx + thidx] = X[thidx + currow + (chunk + thidy)*rows];	
	  __syncthreads();
	  // get maximum in chunk ...
  	  if( thidy == 0 && thidx + currow < rows ){
	      sh_max[thidx] = shared_data[thidx];
	      for( int i = 1; i < bdx; i++){
	      	   if(chunk + i < cols){
	      	      cur_val = sh_max[thidx];
	      	      sh_max[thidx]  = fmax(cur_val, shared_data[i*bdx + thidx]);
                   }
	      }
	      // save values
	      cur_val = y[currow+thidx];
	      y[currow+thidx] = fmax(cur_val, sh_max[thidx]);
	  }
	  __syncthreads();
  }

}

