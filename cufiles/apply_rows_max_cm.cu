
#include <hip/hip_runtime.h>

__global__ void
apply_rows_max_cm(float* X, /** matrix to apply ... column major **/
	               	      float* y, /** result vector  **/
			      int rows,
			      int cols
  ) {

  unsigned int thidx = threadIdx.x;
  unsigned int thidy = threadIdx.y;
  unsigned int bid = blockIdx.x;
  unsigned int bdx = blockDim.x; // assumed equal to blockDim.y .. 16 or 32 ..

  int currow = bdx*bid;

  // flexible block size 
  extern __shared__ float shared_data[];
  float *sh_max = shared_data + bdx*bdx;

  // initialize ...
  if( thidy == 0 && thidx + currow < rows ){
      y[currow+thidx] = -1e37;
      sh_max[thidx] = -1e37;
  }
  __syncthreads();  

  float cur_val;
  for(int chunk = 0; chunk < cols; chunk+=bdx){
  	  // get some values chunking accross rows ...
	  if( thidx + currow < rows && chunk + thidy < cols ){
	      shared_data[thidy*bdx + thidx] = X[thidx + currow + (chunk + thidy)*rows];	
	  }
	  __syncthreads();
	  // get maximum in chunk ...
  	  if( thidy == 0 && thidx + currow < rows ){
	      for( int i = 0; i < bdx; i++){
	      	   if(chunk + i < cols){
	      	      cur_val = sh_max[thidx];
	      	      sh_max[thidx] = fmax(cur_val, shared_data[i*bdx + thidx]);
                   }
	      }
	  }
	  __syncthreads();
  }

  // save results
  if(thidx + currow < rows && thidy==0){
    y[currow + thidx] = sh_max[thidx];}

}

