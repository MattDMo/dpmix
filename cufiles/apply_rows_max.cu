
#include <hip/hip_runtime.h>

__global__ void
apply_rows_max(float* X, /** matrix to apply .. row major **/
	               	      float* y, /** result vector  **/
			      int rows,
			      int cols
  ) {

  unsigned int thidx = threadIdx.x;
  unsigned int thidy = threadIdx.y;
  unsigned int bid = blockIdx.x;
  unsigned int bdx = blockDim.x; // assumed equal to blockDim.y .. 16 or 32 ..

  int currow = bdx*bid;

  // flexible block size 
  extern __shared__ float shared_data[];
  float *sh_max = shared_data + bdx*bdx;

  if( thidy == 0 && thidx + currow < rows  ){
      y[currow+thidx] = -1e37;
      sh_max[thidx] = -1e37;
  }
  __syncthreads();  
  
  float cur_val;
  for(int chunk = 0; chunk < cols; chunk+=bdx){
  	  // get some values chunking accross rows ...
	  if(currow+thidy < rows && chunk + thidx < cols){
	  	shared_data[thidx*bdx + thidy] = X[(currow + thidy)*cols + chunk + thidx];}
	  __syncthreads();
	  // get maximum in chunk ...
  	  if( thidy == 0 && thidx + currow < rows ){
	      for( int i = 0; i < bdx; i++){
	      	   if(chunk + i < cols){
	      	      cur_val = sh_max[thidx];
	      	      sh_max[thidx]  = fmax(cur_val, shared_data[i*bdx + thidx]);
                   }
	      }
	  }
	  __syncthreads();
  }
  // save values
  if(thidx + currow < rows && thidy==0){
    y[currow+thidx] = sh_max[thidx];}

}

   